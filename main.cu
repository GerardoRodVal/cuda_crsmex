#include "hip/hip_runtime.h"
// To Compile it
// gcc -m32 -o test.out test.c sac.a
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <ctype.h>
#include <unistd.h>
#include "crsmex.h"
#include <hip/hip_runtime.h>

extern "C"{
#include <sacio.h>
#include <sac.h>
}

/* Define the maximum length of the data array */
#define MAX 100000
#define NSAC 100
#define N_FILENAME 100
#define MAX_PATH 100

char *strstrip(char *s); // Deletes trailing characters when reading filenames. Similar to .rtrip() in Python.
void usage();            // Show usage
void print_array(float **array, int M, int N);
const char CONFIG_FILENAME[]="config.conf";

int
main(int argc, char **argv)
{
  /* Define variables to be used in the call to rsac1() */
  float   yarray[MAX];
  float   beg, del;
  int     nlen, nerr, max = MAX, opt = 0;
  float   *data[NSAC];
  char    kname[ N_FILENAME ] ;
  char    infilename[ N_FILENAME ] ;
  FILE    *fid;
  size_t  len=0;
//  ssize_t read;
  int     count=0;

  char  *line;
  size_t line_size = 100;

  struct config_filter configstruct;
  configstruct = get_config(CONFIG_FILENAME); 

  printf("Low(int)  = %f\n",configstruct.low);
  printf("High(int)  = %f\n",configstruct.high);
  printf("Attenuation(int)  = %f\n",configstruct.attenuation);
  printf("Transition Band(int)  = %f\n",configstruct.transition_band);
  printf("Npoles  = %d\n",configstruct.npoles);
  printf("passes  = %d\n",configstruct.passes);
 
  if( argc == 1 ) {
	usage();
	exit(-1);
  } 
  while((opt = getopt(argc, argv, "f:")) != -1){
	switch(opt){
	      case 'f':
		strncpy(infilename, optarg, MAX_PATH);
		break;
	default:
		fprintf(stderr, "Unknown option %c\n\n",opt);
		usage();
		exit(-1);
        }
  }
  line = (char  *)malloc(line_size    * sizeof(char));
  for (int i=0; i<NSAC; i++)
  	data[i] = (float *)malloc( MAX  * sizeof(float));  

  fid = fopen(infilename,"r");
  if (fid == NULL){
	fprintf(stderr,"Couldn't open file %s\n",infilename);
	exit(-1);
  } 
 //while ((read = getline(&line, &len, fid)) != -1)
 while (getline(&line, &len, fid) != -1)
  {
	line = strstrip(line);
        strcpy ( kname ,line ) ;
        rsac1( kname, yarray, &nlen, &beg, &del, &max, &nerr, strlen( kname ) ) ;
        if ( nerr != 0 ) {
                fprintf(stderr, "Error reading in SAC file: %s\n", kname);
                exit ( nerr ) ;
        }
	else {
    		fprintf(stderr,"Reading SUCCESS: %s\n",kname);
        	fprintf(stderr,"Number of samples read: %d\n\n",nlen);
	}
         /* START - FILTERING */
    /*     Call xapiir ( Apply a IIR Filter ) 
     *        - yarray - Original Data 
     *        - nlen   - Number of points in yarray 
     *        - proto  - Prototype of Filter 
     *                 - SAC_FILTER_BUTTERWORK        - Butterworth 
     *                 - SAC_FILTER_BESSEL            - Bessel 
     *                 - SAC_FILTER_CHEBYSHEV_TYPE_I  - Chebyshev Type I 
     *                 - SAC_FILTER_CHEBYSHEV_TYPE_II - Chebyshev Type II 
     *        - transition_bandwidth (Only for Chebyshev Filter) 
     *                 - Bandwidth as a fraction of the lowpass prototype 
     *                   cutoff frequency 
     *        - attenuation (Only for Chebyshev Filter) 
     *                 - Attenuation factor, equals amplitude reached at 
     *                   stopband egde 
     *        - order  - Number of poles or order of the analog prototype 
     *                   4 - 5 should be ample 
     *                   Cannot exceed 10 
     *        - type   - Type of Filter 
     *                 - SAC_FILTER_BANDPASS 
     *                 - SAC_FILTER_BANDREJECT 
     *                 - SAC_FILTER_LOWPASS 
     *                 - SAC_FILTER_HIGHPASS 
     *        - low    - Low Frequency Cutoff [ Hertz ] 
     *                   Ignored on SAC_FILTER_LOWPASS 
     *        - high   - High Frequency Cutoff [ Hertz ] 
     *                   Ignored on SAC_FILTER_HIGHPASS 
     *        - delta  - Sampling Interval [ seconds ] 
     *        - passes - Number of passes 
     *                 - 1 Forward filter only 
     *                 - 2 Forward and reverse (i.e. zero-phase) filtering 
     */
    xapiir(yarray, nlen, (char *)SAC_BUTTERWORTH, 
           configstruct.transition_band, configstruct.attenuation, 
           configstruct.npoles, 
           (char *)SAC_HIGHPASS, 
           configstruct.low, configstruct.high, 
           del, configstruct.passes);
       /* END */
	memcpy(data[count],yarray,nlen*sizeof(float));
	count++;
  }

print_array(data,count,nlen);
free(*data);
fclose(fid);
if (line)
        free(line);
 
  
  exit(0);
}

// Strips trailing characters
char *strstrip(char *s)
{
        size_t size;
        char *end;

        size = strlen(s);

        if (!size)
                return s;

        end = s + size - 1;
        while (end >= s && isspace(*end))
                end--;
        *(end + 1) = '\0';

        while (*s && isspace(*s))
                s++;

        return s;
}

void usage(){
fprintf(stderr,"\nCUDA CRSMEX   -  Characteristic Repeating Earthquakes Code \n\n");
fprintf(stderr," This program looks for characteristic repeating earthquakes using GPU/CUDA\n");
fprintf(stderr," Required options:\n");
fprintf(stderr,"                 -f  filenames.dat - filenames.dat must containt a list of all files to be analyzed.\n\n");
fprintf(stderr,"        Author: Luis A. Dominguez - ladominguez@ucla.edu\n\n");

}

void print_array(float **array, int M, int N)
{
FILE *fout;
fprintf(stdout,"M = %d\n",M);
fprintf(stdout,"N = %d\n",N);

fout = fopen("data.dat","w");
	for (int i = 0; i < M; i++){
		for (int j = 0; j < N; j++)
			fprintf(fout,"%8.3f ",array[i][j]);
		fprintf(fout,"\n");
	}
fprintf(stdout, "Writing fie data.dat\n");
fclose(fout);
}

