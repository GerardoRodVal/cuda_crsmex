#include "hip/hip_runtime.h"
// To Compile it
// gcc -m32 -o test.out test.c sac.a
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <ctype.h>
#include <unistd.h>
#include "crsmex.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
extern "C"{
#include <sacio.h>
#include <sac.h>
}

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* Define the maximum length of the data array */
#define MAX_ARRAY 100000
#define NSAC 100
#define N_FILENAME 100
#define MAX_PATH 100

#define GRID_SIZE  1
#define BLOCK_SIZE 4

char *strstrip(char *s); // Deletes trailing characters when reading filenames. Similar to .rtrip() in Python.
void usage();            // Show usage
void print_array(float **array, int M, int N);
void check_gpu_card_type(void);
const char CONFIG_FILENAME[]="config.conf";


__device__  void initDeviceVectors(int *vecA, int lL);
__global__  void find_repeaters(float *data, int npts);

int main(int argc, char **argv)
{
  /* Define variables to be used in the call to rsac1() */
  float     yarray[MAX_ARRAY];
  float     beg, del;
  int       nlen, nerr, max = MAX_ARRAY, opt = 0;
  float     *data[NSAC];
  char      kname[ N_FILENAME ] ;
  char      infilename[ N_FILENAME ] ;
  FILE      *fid;
  size_t    len=0;
  int       count=0;
  hipfftReal *device_data;

  char      *line;
  size_t    line_size = 100;

  /* Filtering variables */
  struct config_filter configstruct;
  configstruct = get_config(CONFIG_FILENAME); 

  /* CUDA configuration */

  int grdSize   = GRID_SIZE;
  int blockSize = BLOCK_SIZE;

  dim3 dimGrid(grdSize, grdSize, grdSize);
  dim3 dimBlock(blockSize, blockSize, blockSize);
   


 /*
  printf("Low(int)  = %f\n",configstruct.low);
  printf("High(int)  = %f\n",configstruct.high);
  printf("Attenuation(int)  = %f\n",configstruct.attenuation);
  printf("Transition Band(int)  = %f\n",configstruct.transition_band);
  printf("Npoles  = %d\n",configstruct.npoles);
  printf("passes  = %d\n",configstruct.passes);
 */

  if( argc == 1 ) {
	usage();
	exit(-1);
  }

  // Check is a GPU card is available.
  check_gpu_card_type();

  // Retrieve input parameters 
  while((opt = getopt(argc, argv, "f:")) != -1){
	switch(opt){
	      case 'f':
		strncpy(infilename, optarg, MAX_PATH);
		break;
	default:
		fprintf(stderr, "Unknown option %c\n\n",opt);
		usage();
		exit(-1);
        }
  }

  line = (char  *)malloc(line_size    * sizeof(char));

  for (int i=0; i<NSAC; i++)
  	data[i] = (float *)malloc( MAX_ARRAY  * sizeof(float));  

  // Read input filenames.
  fid = fopen(infilename,"r");
  if (fid == NULL){
	fprintf(stderr,"Couldn't open file %s\n",infilename);
	exit(-1);
  } 
 

 // Read sac files into host memory.
 while (getline(&line, &len, fid) != -1)
  {
	line = strstrip(line);
        strcpy ( kname ,line ) ;
        rsac1( kname, yarray, &nlen, &beg, &del, &max, &nerr, strlen( kname ) ) ;
        if ( nerr != 0 ) {
                fprintf(stderr, "Error reading in SAC file: %s\n", kname);
                exit ( nerr ) ;
        }
	else {
    		fprintf(stderr,"Reading SUCCESS: %s\n",kname);
        	fprintf(stderr,"Number of samples read: %d\n\n",nlen);
	}
         /* START - FILTERING */
    /*     Call xapiir ( Apply a IIR Filter ) 
     *        - yarray - Original Data 
     *        - nlen   - Number of points in yarray 
     *        - proto  - Prototype of Filter 
     *                 - SAC_FILTER_BUTTERWORK        - Butterworth 
     *                 - SAC_FILTER_BESSEL            - Bessel 
     *                 - SAC_FILTER_CHEBYSHEV_TYPE_I  - Chebyshev Type I 
     *                 - SAC_FILTER_CHEBYSHEV_TYPE_II - Chebyshev Type II 
     *        - transition_bandwidth (Only for Chebyshev Filter) 
     *                 - Bandwidth as a fraction of the lowpass prototype 
     *                   cutoff frequency 
     *        - attenuation (Only for Chebyshev Filter) 
     *                 - Attenuation factor, equals amplitude reached at 
     *                   stopband egde 
     *        - order  - Number of poles or order of the analog prototype 
     *                   4 - 5 should be ample 
     *                   Cannot exceed 10 
     *        - type   - Type of Filter 
     *                 - SAC_FILTER_BANDPASS 
     *                 - SAC_FILTER_BANDREJECT 
     *                 - SAC_FILTER_LOWPASS 
     *                 - SAC_FILTER_HIGHPASS 
     *        - low    - Low Frequency Cutoff [ Hertz ] 
     *                   Ignored on SAC_FILTER_LOWPASS 
     *        - high   - High Frequency Cutoff [ Hertz ] 
     *                   Ignored on SAC_FILTER_HIGHPASS 
     *        - delta  - Sampling Interval [ seconds ] 
     *        - passes - Number of passes 
     *                 - 1 Forward filter only 
     *                 - 2 Forward and reverse (i.e. zero-phase) filtering 
     */
    xapiir(yarray, nlen, (char *)SAC_BUTTERWORTH, 
           configstruct.transition_band, configstruct.attenuation, 
           configstruct.npoles, 
           (char *)SAC_HIGHPASS, 
           configstruct.low, configstruct.high, 
           del, configstruct.passes);
       /* END */
	memcpy(data[count],yarray,nlen*sizeof(float));
	count++;
  }

  /* CUDA FFT */
  hipfftHandle plan;
  hipfftComplex *fft_data;
  int rank = 1;                                  // --- 1D FFTs
  int n[] = { nlen };                            // --- Size of the Fourier transform
  int istride = 1, ostride = 1;                  // --- Distance between two successive input/output elements
  int idist = MAX_ARRAY, odist = (nlen / 2 + 1); // --- Distance between batches
  int inembed[] = { 0 };                         // --- Input size with pitch (ignored for 1D transforms)
  int onembed[] = { 0 };                         // --- Output size with pitch (ignored for 1D transforms)
  int batch = count;                             // --- Number of batched executions

 
  // Initiazilizing device data for fft processing
  gpuErrchk(hipMalloc((void**)&device_data,    MAX_ARRAY * count * sizeof(hipfftReal   )));
  gpuErrchk(hipMalloc((void**)&fft_data,  (nlen / 2 + 1) * count * sizeof(hipfftComplex)));

  hipMemcpy(data,device_data, count * nlen * sizeof(float), hipMemcpyHostToDevice);

  hipfftPlanMany(&plan, rank, n, 
                inembed, istride, idist,
                onembed, ostride, odist, HIPFFT_R2C, batch);
   hipfftExecR2C(plan, device_data, fft_data);
  //gpuErrchk(hipMemcpy(device_data, data, count*nlen*sizeof(float), hipMemcpyHostToDevice));
/*
  hipMemcpy2DToArray(device_data, 
                    0, 
                    0,  
                    data,
                    MAX_ARRAY * sizeof(float),  
                    nlen      * sizeof(float), 
                    count     * sizeof(float),  hipMemcpyHostToDevice);
  printf("Hola\n");
*/
printf("idist = %d\n", idist);
printf("odist = %d\n", odist);
printf("n = %d\n", n[0]);

find_repeaters<<<count, nlen >>> (device_data, nlen);


gpuErrchk(hipFree(device_data));

//print_array(data,count,nlen);
free(*data);
fclose(fid);
if (line)
        free(line);
 
hipDeviceReset();  
return EXIT_SUCCESS;
}

__global__ void find_repeaters(float *data,int npts){
__shared__ float* trace;

trace = (float *)malloc(npts*sizeof(float));

for(int currentBlockOfPoints = 0; currentBlockOfPoints < gridDim.x; currentBlockOfPoints++)
	trace[threadIdx.x] = data[threadIdx.x + currentBlockOfPoints*npts];
	
}

// Strips trailing characters
char *strstrip(char *s)
{
        size_t size;
        char *end;

        size = strlen(s);

        if (!size)
                return s;

        end = s + size - 1;
        while (end >= s && isspace(*end))
                end--;
        *(end + 1) = '\0';

        while (*s && isspace(*s))
                s++;

        return s;
}

void usage(){
fprintf(stderr,"\nCUDA CRSMEX   -  Characteristic Repeating Earthquakes Code \n\n");
fprintf(stderr," This program looks for characteristic repeating earthquakes using GPU/CUDA\n");
fprintf(stderr," Required options:\n");
fprintf(stderr,"                 -f  filenames.dat - filenames.dat must containt a list of all files to be analyzed.\n\n");
fprintf(stderr,"        Author: Luis A. Dominguez - ladominguez@ucla.edu\n\n");

}

void print_array(float **array, int M, int N)
{
FILE *fout;
fprintf(stdout,"M = %d\n",M);
fprintf(stdout,"N = %d\n",N);

fout = fopen("data.dat","w");
	for (int i = 0; i < M; i++){
		for (int j = 0; j < N; j++)
			fprintf(fout,"%8.3f ",array[i][j]);
		fprintf(fout,"\n");
	}
fprintf(stdout, "Writing fie data.dat\n");
fclose(fout);
}

__device__ void initDeviceVectors(int *vecA, int lL){

	
}

void check_gpu_card_type()
{
  int nDevices;
  hipGetDeviceCount(&nDevices);
  if (nDevices == 0){
	fprintf(stderr,"ERROR - No GPU card detected.\n");
	exit(-1);
  }

  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}

