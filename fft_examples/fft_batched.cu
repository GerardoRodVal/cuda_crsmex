//nvcc -arch=sm_30 -lcufft fft_batched.cu


#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <math.h>

#define DATASIZE 8
#define BATCH 3

#define GRID_DIMENSION  3
#define BLOCK_DIMENSION 3



/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
__global__ void conjugate(long int nelem, hipfftComplex *conj);


/********/
/* MAIN */
/********/
int main ()
{
    
    // --- Host side input data allocation and initialization
    hipfftReal *hostInputData = (hipfftReal*)malloc(DATASIZE*BATCH*sizeof(hipfftReal));
    int grid_size  = GRID_DIMENSION;
    int block_size = BLOCK_DIMENSION;

    dim3 DimGrid(grid_size, grid_size, grid_size);
    dim3 DimBlock(block_size, block_size, block_size);


    for (int i=0; i<BATCH; i++)
        for (int j=0; j<DATASIZE; j++){ 
		hostInputData[i*DATASIZE + j] = (hipfftReal)((i + 1) + j);
		printf("hostInputData[%d]=%f\n",i*DATASIZE + j,hostInputData[i*DATASIZE + j]);
	}

    // --- Device side input data allocation and initialization
    hipfftReal *deviceInputData; 
    gpuErrchk(hipMalloc((void**)&deviceInputData, DATASIZE * BATCH * sizeof(hipfftReal)));

    hipMemcpy(deviceInputData, hostInputData, DATASIZE * BATCH * sizeof(hipfftReal), hipMemcpyHostToDevice);

    // --- Host side output data allocation
    hipfftComplex *hostOutputData = (hipfftComplex*)malloc((DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex));

    // --- Device side output data allocation
    hipfftComplex *deviceOutputData; 
    hipfftComplex *fft_conj; 
    gpuErrchk(hipMalloc((void**)&deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex)));
    gpuErrchk(hipMalloc((void**)&fft_conj,         (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex)));

    // --- Batched 1D FFTs
    hipfftHandle handle;
    int rank = 1;                           // --- 1D FFTs
    int n[] = { DATASIZE };                 // --- Size of the Fourier transform
    int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
    int idist = DATASIZE, odist = (DATASIZE / 2 + 1); // --- Distance between batches
    int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
    int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
    int batch = BATCH;                      // --- Number of batched executions

    printf("idist = %d\n", idist);
    printf("odist = %d\n", odist);
    printf("n = %d\n", n[0]);

    hipfftPlanMany(&handle, rank, n,
                  inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_R2C, batch);

    //cufftPlan1d(&handle, DATASIZE, CUFFT_R2C, BATCH);
    hipfftExecR2C(handle,  deviceInputData, deviceOutputData);
    gpuErrchk(hipMemcpy(fft_conj,       deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex), hipMemcpyDeviceToDevice));
    conjugate <<< DimGrid, DimBlock >>> ((DATASIZE / 2 + 1) * BATCH, fft_conj );
    // --- Device->Host copy of the results
    gpuErrchk(hipMemcpy(hostOutputData, deviceOutputData, (DATASIZE / 2 + 1) * BATCH * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

    for (int i=0; i<BATCH; i++)
        for (int j=0; j<(DATASIZE / 2 + 1); j++)
            printf("Batch  = %i j= %i real %f imag %f\n", i, j, hostOutputData[i*(DATASIZE / 2 + 1) + j].x, hostOutputData[i*(DATASIZE / 2 + 1) + j].y);

    hipfftDestroy(handle);
    gpuErrchk(hipFree(deviceOutputData));
    gpuErrchk(hipFree(deviceInputData));
    gpuErrchk(hipFree(fft_conj));
    hipDeviceSynchronize();
    hipDeviceReset();
    return EXIT_SUCCESS;

}

__global__ void conjugate(long int nelem, hipfftComplex *conj)
{
int bx = blockIdx.x;
int by = blockIdx.y;
int bz = blockIdx.z;

int thx = threadIdx.x;
int thy = threadIdx.y;
int thz = threadIdx.z;

int NumThread = blockDim.x*blockDim.y*blockDim.z;
int idThread  = (thx + thy*blockDim.x) + thz*(blockDim.x*blockDim.y);
int BlockId   = (bx + by*gridDim.x) + bz*(gridDim.x*gridDim.y);

int uniqueid  = idThread + NumThread*BlockId;
if (uniqueid < nelem){
 	printf("Unique ID = %d - conj = %f\n",  uniqueid,  conj[uniqueid].y*-1);
}

//__syncthreads();
}
